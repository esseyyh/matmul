#include <hip/hip_runtime.h>
#include <float.h>
#include <stdio.h>
#include <stdlib.h>
#include <time.h>

#define NUM_REPETITIONS 5
#define TILE_SIZE 32
__global__ void matmul_kernel_tiled(float *A, float *B, float *C, int M, int N,
                                    int K) {
  __shared__ float A_shared[TILE_SIZE][TILE_SIZE];
  __shared__ float B_shared[TILE_SIZE][TILE_SIZE];

  int bx = blockIdx.x;
  int by = blockIdx.y;
  int tx = threadIdx.x;
  int ty = threadIdx.y;

  int row = by * TILE_SIZE + ty;
  int col = bx * TILE_SIZE + tx;

  float sum = 0.0f;

  for (int tile = 0; tile < (K + TILE_SIZE - 1) / TILE_SIZE; ++tile) {
    if (row < M && tile * TILE_SIZE + tx < K) {
      A_shared[ty][tx] = A[row * K + tile * TILE_SIZE + tx];
    } else {
      A_shared[ty][tx] = 0.0f;
    }

    if (col < N && tile * TILE_SIZE + ty < K) {
      B_shared[ty][tx] = B[(tile * TILE_SIZE + ty) * N + col];
    } else {
      B_shared[ty][tx] = 0.0f;
    }

    __syncthreads();

    float4 sum4 = make_float4(0.0f, 0.0f, 0.0f, 0.0f);

    for (int k = 0; k < TILE_SIZE; k += 4) {
      float4 a_vec = *reinterpret_cast<float4 *>(&A_shared[ty][k]);
      float4 b_vec = make_float4(B_shared[k][tx], B_shared[k + 1][tx],
                                 B_shared[k + 2][tx], B_shared[k + 3][tx]);

      sum4.x += a_vec.x * b_vec.x;
      sum4.y += a_vec.y * b_vec.y;
      sum4.z += a_vec.z * b_vec.z;
      sum4.w += a_vec.w * b_vec.w;
    }

    sum += sum4.x + sum4.y + sum4.z + sum4.w;
    __syncthreads();
  }

  if (row < M && col < N) {
    C[row * N + col] = sum;
  }
}

__global__ void matmul_kernel(float *A, float *B, float *C, int M, int N,
                              int K) {
  int row = blockIdx.y * blockDim.y + threadIdx.y;
  int col = blockIdx.x * blockDim.x + threadIdx.x;
  if (row < M && col < N) {
    float sum = 0.0f;
    for (int k = 0; k < K; ++k) {
      sum += A[row * K + k] * B[k * N + col];
    }
    C[row * N + col] = sum;
  }
}

void matmul(float *A, float *B, float *C, int M, int N, int K) {
  // float *d_A, *d_B, *d_C;

  int M_per_gpu = M / 2;

  float *d_A[2], *d_B[2], *d_C[2];

  for (int gpu = 0; gpu < 2; ++gpu) {
    hipSetDevice(gpu);

    hipMalloc(&d_A[gpu], M_per_gpu * K * sizeof(float));
    hipMalloc(&d_B[gpu], K * N * sizeof(float));
    hipMalloc(&d_C[gpu], M_per_gpu * N * sizeof(float));

    hipMemcpy(d_A[gpu], A + gpu * M_per_gpu * K, M_per_gpu * K * sizeof(float),
               hipMemcpyHostToDevice);
    hipMemcpy(d_B[gpu], B, K * N * sizeof(float), hipMemcpyHostToDevice);
  }

  dim3 blockDim(16, 16);
  dim3 gridDim((N + 15) / 16, (M_per_gpu + 15) / 16);
  for (int gpu = 0; gpu < 2; ++gpu) {
    hipSetDevice(gpu);
    matmul_kernel_tiled<<<gridDim, blockDim>>>(d_A[gpu], d_B[gpu], d_C[gpu],
                                               M_per_gpu, N, K);
  }

  for (int gpu = 0; gpu < 2; ++gpu) {
    hipSetDevice(gpu);
    hipMemcpy(C + gpu * M_per_gpu * N, d_C[gpu], M_per_gpu * N * sizeof(float),
               hipMemcpyDeviceToHost);

    hipFree(d_A[gpu]);
    hipFree(d_B[gpu]);
    hipFree(d_C[gpu]);
  }

  hipFree(d_A);
  hipFree(d_B);
  hipFree(d_C);
}

double get_time() {
  struct timespec ts;
  clock_gettime(CLOCK_MONOTONIC, &ts);
  return ts.tv_sec + ts.tv_nsec * 1e-9;
}

int main(int argc, char *argv[]) {
  int sizes[][3] = {{128, 128, 128},      {512, 512, 512},
                    {1024, 1024, 1024},   {2048, 2048, 2048},
                    {4096, 4096, 4096},   {8192, 8192, 8192},
                    {16384, 16384, 16384}};
  int num_sizes = sizeof(sizes) / sizeof(sizes[0]);

  srand(time(NULL));

  printf("m,n,k,time,gflops\n");

  double best_gflops = 0.0;
  int best_m = 0, best_n = 0, best_k = 0;

  for (int i = 0; i < num_sizes; i++) {
    int M = sizes[i][0];
    int N = sizes[i][1];
    int K = sizes[i][2];
    float *A = (float *)malloc(M * K * sizeof(float));
    float *B = (float *)malloc(K * N * sizeof(float));
    float *C = (float *)malloc(M * N * sizeof(float));
    if (!A || !B || !C) {
      fprintf(stderr, "Memory allocation failed\n");
      exit(1);
    }

    for (int j = 0; j < M * K; j++) {
      A[j] = (float)rand() / RAND_MAX;
    }
    for (int j = 0; j < K * N; j++) {
      B[j] = (float)rand() / RAND_MAX;
    }

    double total_time = 0.0;
    double min_time = DBL_MAX;

    for (int rep = 0; rep < NUM_REPETITIONS; rep++) {
      double start_time = get_time();
      matmul(A, B, C, M, N, K);
      double end_time = get_time();
      double elapsed_time = end_time - start_time;

      total_time += elapsed_time;
      if (elapsed_time < min_time) {
        min_time = elapsed_time;
      }
    }

    double avg_time = total_time / NUM_REPETITIONS;
    double flops = 2.0 * M * N * K;
    double avg_gflops = flops / (avg_time * 1e9);
    double max_gflops = flops / (min_time * 1e9);

    printf("%d,%d,%d,%.6f,%.2f\n", M, N, K, avg_time, avg_gflops);

    if (max_gflops > best_gflops) {
      best_gflops = max_gflops;
      best_m = M;
      best_n = N;
      best_k = K;
    }

    free(A);
    free(B);
    free(C);
  }

  printf("\nBest configuration:\n");
  printf("M=%d, N=%d, K=%d\n", best_m, best_n, best_k);
  printf("Best performance: %.2f GFLOPS\n", best_gflops);

  return 0;
}
